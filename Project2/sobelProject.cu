#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <ctime>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
// Waris     Vorathumdusadee Section 3 6088128
// Sirichoke Yooyen          Section 3 6088232
// Phummarat Yosamornsuntorn Section 3 608823

// this is the kernel for change RGB to grey color
__global__ void ChangeToGrey(unsigned char *rgb, unsigned char *grey, int Row, int Column) {

    //find the global ID of each row
    int ColIDThread = threadIdx.x + blockIdx.x * blockDim.x;
    //find the global ID of each columns
    int RowIDThread = threadIdx.y + blockIdx.y * blockDim.y;


          int NumberOffset = RowIDThread * Column + ColIDThread;
          //multiply 3 because 3 color which is r g b
          int rgb_offset = NumberOffset * 3;
          //
          float weight[3] = {0.22,0.72,0.07};
          // this is the weight that to compute the gray scale
          float sum=0; //sum for compute the value that of all RGB color

          //this loop for find the value of Gray color
          for(int i=0;i<3 ;i++)
          {
            sum += rgb[rgb_offset+i]*weight[i];
          }
          grey[NumberOffset] = sum;

}
// this is the kernel for find the sobel of X and Y
__global__ void ChangeToSobel(unsigned char *grayData,
  unsigned char *ValueSobelX,
  unsigned char *ValueSobelY,
  int Row,
  int Column) {
    //find the global ID of each columns
    int ColIDThread = threadIdx.x + blockIdx.x * blockDim.x;
    //find the global ID of each row
    int RowIDThread = threadIdx.y + blockIdx.y * blockDim.y;
        // calculate Global threadID
        int offset = RowIDThread * Column + ColIDThread;
        int ValueOfMargin[3][3];  //it is used for save the value of margin of that pixel
        // set the matrix value for find the sobel of horizontal
        int Horizon[3][3] = {{1,0,-1},{2,0,-2},{1,0,-1}};
        // set the matrix value for find the sobel of vertical
        int Vertic[3][3] = {{1,2,1},{0,0,0},{-1,-2,-1}};

        int step = -1; //step for find the neighbor
        // this loop for find the neighbor of that pixel
        for(int i=0;i<3;i++)
        {
          for(int j=0;j<3;j++)
          {

              ValueOfMargin[i][j] = (RowIDThread+(i-1))*Column + (ColIDThread+step);
              //set the value of margin of that pixel
              step++;
              // increase because use for go to the pixel
          }
          step=-1;
        }
        int sumHorizon = 0;// set the matrix value for find the sobel of horizontal
        int sumVertical = 0;// set the matrix value for find the sobel of vertical
        // this loop for find the sobel in the vertical and horizontal
        for(int i = 0; i < 3; i ++){
            for(int j = 0; j < 3; j++){

                int TempX = sumHorizon;
                // TempX for save the previous value of horizontal
                int TempY = sumVertical;
                // TempY for save the previous value of vertical
                sumVertical = (grayData[ValueOfMargin[i][j]] * Horizon[i][j]);
                sumVertical=sumVertical+TempY;
                // set the new value sum of vertical
                sumHorizon = (grayData[ValueOfMargin[i][j]] * Vertic[i][j]);
                sumHorizon = TempX+sumHorizon;
                  // set the new value sum of horizontal
            }

        }
          // set the value sum of horizontal if it more than 255 set it to 255
          // because the value cannot exceed 255
        if(sumHorizon >= 255){
            sumHorizon = 255;
        }
        // set the value sum of horizontal if it less than 0 set it to 0
        // because the value cannot less than 0
        if(sumHorizon <= 0){
            sumHorizon = 0;
        }
        // set the value sum of vertical if it more than 255 set it to 255
        // because the value cannot exceed 255
        if(sumVertical >= 255){
            sumVertical = 255;
        }
        // set the value sum of vertical if it less than 0 set it to 0
        // because the value cannot less than 0
        if(sumVertical <= 0){
            sumVertical = 0;
        }
        //set the value of sobelX
        ValueSobelX[offset] = sumHorizon;
        //set the value of sobelY
        ValueSobelY[offset] = sumVertical;

}

int main(int argc, char **argv) {

    clock_t begin = clock();
	  std::string input_file;
    std::string OutputFilename[3];
    // OutputFilename[0] for gray image output
    // OutputFilename[1] for sobelX of image output
    // OutputFilename[2] for sobelY of image output

	//Check for the input file that user input or not
      if(argc==1)
    {
      std::cerr << "Please input your image file in the directory first.\n";
        std::cerr << "And type ./sobelProject filename.jpg\n";
      exit(1);
    }
    // set the value of output file use for write the output to the user
			      input_file = std::string(argv[1]);
            OutputFilename[0] = "Gray_Of_"+std::string(argv[1]);
            OutputFilename[1] = "OutSobelX_Of_"+std::string(argv[1]);
            OutputFilename[2] = "OutSobelY_Of_"+std::string(argv[1]);
    unsigned char *DataForCopy[4];
    // That use for cuda to do in the device
    // DataForCopy[0] for array for storing rgb data
    //DataForCopy[1] for data of grey
    //DataForCopy[2] for data of sobelX
    //DataForCopy[3] for data of sobelY

    unsigned char *Data[3];
      // That use for host
    //Data[0] for data of grey
    //Data[1] for data of sobelX
    //Data[2] for data of sobelY


  	int RowOfPix; //number of rows of pixels
  	int ColumnfPix; //number of columns of pixels

    //read image data into DataImg Mat object
    cv::Mat DataImg = cv::imread(input_file, cv::IMREAD_ANYCOLOR | cv::IMREAD_ANYDEPTH);
    if (DataImg.empty()) {
        std::cerr << "Please check your " << input_file << std::endl;
        std::cerr << "It is not in the same directory of .cu "<< std::endl;
    }
    // set the value of RowOfPix equal the row of image
    RowOfPix = DataImg.rows;
    // set the value of ColumnfPix equal the colums of image
    ColumnfPix = DataImg.cols;

    // find the total size of image
    const size_t SizeofAllPixel = DataImg.rows * DataImg.cols;
    // find the value of all pixel that contains RGB color
    const size_t SizeofAllRGB = 3*SizeofAllPixel;

	//this loop for allocate the memory
    for(int i=0;i<4;i++)
    {
      if(i==0)
      {
        //allocate for
        //DataForCopy[0] for array for storing rgb data
         hipMalloc(&DataForCopy[i], sizeof(unsigned char)* SizeofAllRGB);
      }
      else
      {
         //allocate for
         //DataForCopy[1] for data of greydata
         //DataForCopy[2] for data of sobelX
         //DataForCopy[3] for data of sobelY
         //Data[0] for data of grey
         //Data[1] for data of sobelX
         //Data[2] for data of sobelY

        Data[i-1] = (unsigned char *)malloc(sizeof(unsigned char*)* SizeofAllPixel);
         hipMalloc(&DataForCopy[i], sizeof(unsigned char) * SizeofAllPixel);
      }
    }
  hipMemset(DataForCopy[1], 0, sizeof(unsigned char) * SizeofAllPixel);

	//use hipMemcpy for copy data from host to device
    hipMemcpy(DataForCopy[0], (unsigned char*)DataImg.data, sizeof(unsigned char) *SizeofAllRGB, hipMemcpyHostToDevice);
    hipMemcpy(DataForCopy[2], Data[1], sizeof(unsigned char) * SizeofAllPixel, hipMemcpyHostToDevice);
    hipMemcpy(DataForCopy[3], Data[2], sizeof(unsigned char) * SizeofAllPixel, hipMemcpyHostToDevice);


  int Thread = 16; //maximum thread
	const dim3 dimBlock(Thread, Thread); //16*16 thread per blocks
  int x=ColumnfPix%Thread;
  int y=RowOfPix%Thread;
  // below condition for n divide by number of thread and remainder equal 0 or not
  if(x==0 && y==0)
  {

    	//this the cuda kernal and we set the dimGrid and dimBlock
      const dim3 dimGrid((ColumnfPix)/Thread, (RowOfPix)/Thread); //number of block
      ChangeToGrey<<<dimGrid, dimBlock>>>(DataForCopy[0], DataForCopy[1], RowOfPix, ColumnfPix);
      ChangeToSobel<<<dimGrid, dimBlock>>>(DataForCopy[1],DataForCopy[2], DataForCopy[3], RowOfPix, ColumnfPix);
  }
  if(x!=0 && y!=0)
  {

    	//this the cuda kernal and we set the dimGrid and dimBlock
      const dim3 dimGrid((int)ceil((ColumnfPix+Thread-1)/Thread), (RowOfPix+Thread-1)/Thread);//number of block
      ChangeToGrey<<<dimGrid, dimBlock>>>(DataForCopy[0], DataForCopy[1], RowOfPix, ColumnfPix);
      ChangeToSobel<<<dimGrid, dimBlock>>>(DataForCopy[1],DataForCopy[2], DataForCopy[3], RowOfPix, ColumnfPix);
  }
  if(x!=0 && y==0)
  {

    	//this the cuda kernal and we set the dimGrid and dimBlock
      const dim3 dimGrid((ColumnfPix+Thread-1)/Thread, (RowOfPix)/Thread);//number of block
      ChangeToGrey<<<dimGrid, dimBlock>>>(DataForCopy[0], DataForCopy[1], RowOfPix, ColumnfPix);
      ChangeToSobel<<<dimGrid, dimBlock>>>(DataForCopy[1],DataForCopy[2], DataForCopy[3], RowOfPix, ColumnfPix);
  }
  if(x!=0 && y!=0)
  {

    	//this the cuda kernal and we set the dimGrid and dimBlock
      const dim3 dimGrid((ColumnfPix)/Thread, (RowOfPix+Thread-1)/Thread);//number of block
      ChangeToGrey<<<dimGrid, dimBlock>>>(DataForCopy[0], DataForCopy[1], RowOfPix, ColumnfPix);
      ChangeToSobel<<<dimGrid, dimBlock>>>(DataForCopy[1],DataForCopy[2], DataForCopy[3], RowOfPix, ColumnfPix);
  }




	//copy computed gray data array from device to host
  for(int i=0;i<3;i++)
  {
    	//copy DataForCopy to Data by using hipMemcpyDeviceToHost
      hipMemcpy(Data[i], DataForCopy[i+1], sizeof(unsigned char) * SizeofAllPixel, hipMemcpyDeviceToHost);
      	//set the Mat to save the value of data
        cv::Mat outputData(RowOfPix, ColumnfPix, CV_8UC1,(void *) Data[i]);
        // write the file by using above Mat
          cv::imwrite(OutputFilename[i].c_str(), outputData);
          // print to the user that it write file already
          std::cout <<"Write file " << OutputFilename[i] << std::endl;
  }
  // this loop for clear memory in that store in the array
  for(int i=0;i<4;i++)
  {
     hipFree(DataForCopy[i]);
  }
  clock_t finish = clock();
  double timeuse = double(finish - begin);
  fflush(stdout);
  //print the time that use to execute
  std::cout <<"Time used: " << timeuse << std::endl;
	return 0;
}
