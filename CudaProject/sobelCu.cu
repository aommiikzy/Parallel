#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <ctime>

// store data of image
unsigned char *data_rgb_image;

// cuda kernel
__global__ void RGBtoGrey(unsigned char *rgb, unsigned char *grey, int _row, int _col) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    // calculate for only in the GridDim
    if (col < _col && row < _row) {
        // calculate Global threadID
      const int grey_offset = row * _col + col;
        // 3 mean channel (R G B)
      const int rgb_offset = grey_offset * 3;
        // use unsigned char because size of it is 0 - 255 that mach with color (0-255)
    	const unsigned char r = rgb[rgb_offset + 0]; // 0 for red
	    const unsigned char g = rgb[rgb_offset + 1]; // 1 for green
	    const unsigned char b = rgb[rgb_offset + 2]; // 2 for blue

	    grey[grey_offset] = r * 0.21f + g * 0.71f + b * 0.07f;
    }
}

__global__ void GreytoSobel(unsigned char *gray, unsigned char *sobelX, unsigned char *sobelY, int _row, int _col) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    if ( col< _col && row < _row) {
        // calculate Global threadID
        int sobel_offset = row * _col + col;
        int margin[3][3];
        // Gx for mask of horizontal
        int Gx[3][3] = {{1,0,-1},{2,0,-2},{1,0,-1}};
        // Gy for mask of vertical
        int Gy[3][3] = {{1,2,1},{0,0,0},{-1,-2,-1}};
        //calculate margin for 3 x 3 matrix (neighbor of pixel)
        margin[0][0] = (row-1) * _col + col - 1;
        margin[0][1] = (row-1) * _col + col;
        margin[0][2] = (row-1) * _col + col + 1;
        margin[1][0] = row * _col + col - 1;
        margin[1][1] = row * _col + col;
        margin[1][2] = row * _col + col + 1;
        margin[2][0] = (row+1) * _col + col - 1;
        margin[2][1] = (row+1) * _col + col;
        margin[2][2] = (row+1) * _col + col + 1;

        int sumX = 0, sumY = 0;
        // calcurate sobel
        for(int i = 0; i < 3; i ++){
            for(int j = 0; j < 3; j++){
                sumX += gray[margin[i][j]] * Gx[i][j];
                sumY += gray[margin[i][j]] * Gy[i][j];
            }
        }
        if(sumX >= 255){
            sumX = 255;
        }
        if(sumY >= 255){
            sumY = 255;
        }
        if(sumX <= 0){
            sumX = 0;
        }
        if(sumY <= 0){
            sumY = 0;
        }
        // printf("%d\n", sumX);
        sobelX[sobel_offset] = sumX;
        sobelY[sobel_offset] = sumY;
    }
}
//function for loading an image into rgb format unsigned char array
size_t loadImgFile(unsigned char *grey_image, const std::string &input_file, int *rows, int *cols) {

    cv::Mat img_data; //opencv Mat object
    //read image data into img_data Mat object
    img_data = cv::imread(input_file.c_str(), cv::IMREAD_ANYCOLOR | cv::IMREAD_ANYDEPTH);
    if (img_data.empty()) {
        std::cerr << "Unable to laod image file: " << input_file << std::endl;
    }

    *rows = img_data.rows;
    *cols = img_data.cols;

    //allocate memory for host rgb data array
    data_rgb_image = (unsigned char*) malloc(*rows * *cols * sizeof(unsigned char) * 3);
    unsigned char* rgb_image = (unsigned char*)img_data.data;

    //populate host's rgb data array
    int x = 0;
    for (x = 0; x < *rows * *cols * 3; x++) {
        data_rgb_image[x] = rgb_image[x];
    }
    // return size of image in pixel
    return img_data.rows * img_data.cols;
}

// method for writing image file of gray scale
void outputImg(const std::string& output_file, unsigned char* grey_image, int _row, int _col) {

	//serialize gray data array into opencv's Mat object
	cv::Mat greyData(_row, _col, CV_8UC1,(void *) grey_image);
	//write Mat object to file
    cv::imwrite(output_file.c_str(), greyData);
    printf("Transfer complete\n");
}

// size_t (unsigned integer)
size_t loadImgFile(unsigned char *grey_img, const std::string &input_file, int *_row, int *_col );
void outputImg(const std::string &output_file, unsigned char *grey_image, int _row, int _col);

int main(int argc, char **argv) {

    clock_t begin = clock();
	  std::string input_file;
    std::string gray_file;
    std::string horizontal_sobel_file;
    std::string vertical_sobel_file;

	//Check for the input file and output file names
	switch(argc) {
		case 5:
			      input_file = std::string(argv[1]);
            gray_file = std::string(argv[2]);
            horizontal_sobel_file = std::string(argv[3]);
            vertical_sobel_file = std::string(argv[4]);
            break;
		default:
			std::cerr << "Usage: <executable> input_file gray_file horizontal_sobel_file vertical_sobel_file\n";
			exit(1);
	}

	unsigned char *d_rgb_image; //array for storing rgb data on device
    unsigned char *data_grey_image = 0;
    unsigned char *data_sobel_x_image = 0;
    unsigned char *data_sobel_y_image = 0;
    unsigned char *d_grey_image = 0; //host and device's grey data array pointers
    unsigned char *d_sobel_x = 0;
    unsigned char *d_sobel_y = 0;
	int p_rows; //number of rows of pixels
	int p_cols; //number of columns of pixels

	//load image into an array and retrieve number of pixels
    const size_t total_pixels = loadImgFile(data_grey_image, input_file, &p_rows, &p_cols);

	//allocate memory of host's grey data array
    data_grey_image = (unsigned char *)malloc(sizeof(unsigned char*)* total_pixels);
    data_sobel_x_image = (unsigned char *)malloc(sizeof(unsigned char*)* total_pixels);
    data_sobel_y_image = (unsigned char *)malloc(sizeof(unsigned char*)* total_pixels);

	//allocate and initialize memory on device
	hipMalloc(&d_rgb_image, sizeof(unsigned char) * total_pixels * 3); // 3 is for channel (R G B)
    hipMalloc(&d_grey_image, sizeof(unsigned char) * total_pixels);
    hipMalloc(&d_sobel_x, sizeof(unsigned char) * total_pixels);
    hipMalloc(&d_sobel_y, sizeof(unsigned char) * total_pixels);
	hipMemset(d_grey_image, 0, sizeof(unsigned char) * total_pixels);

	//copy host rgb data array to device data array
    hipMemcpy(d_rgb_image, data_rgb_image, sizeof(unsigned char) * total_pixels * 3, hipMemcpyHostToDevice);
    hipMemcpy(d_sobel_x, data_sobel_x_image, sizeof(unsigned char) * total_pixels, hipMemcpyHostToDevice);
    hipMemcpy(d_sobel_y, data_sobel_y_image, sizeof(unsigned char) * total_pixels, hipMemcpyHostToDevice);

	//define block and grid dimensions
	const dim3 dimGrid((int)ceil((p_cols)/16), (int)ceil((p_rows)/16));
	const dim3 dimBlock(16, 16);

	//execute cuda kernel
    RGBtoGrey<<<dimGrid, dimBlock>>>(d_rgb_image, d_grey_image, p_rows, p_cols);
    GreytoSobel<<<dimGrid, dimBlock>>>(d_grey_image,d_sobel_x, d_sobel_y, p_rows, p_cols);

	//copy computed gray data array from device to host
    hipMemcpy(data_grey_image, d_grey_image, sizeof(unsigned char) * total_pixels, hipMemcpyDeviceToHost);
    hipMemcpy(data_sobel_x_image, d_sobel_x, sizeof(unsigned char) * total_pixels, hipMemcpyDeviceToHost);
    hipMemcpy(data_sobel_y_image, d_sobel_y, sizeof(unsigned char) * total_pixels, hipMemcpyDeviceToHost);

	//output the grayscale image
    outputImg(gray_file, data_grey_image, p_rows, p_cols);
    outputImg(horizontal_sobel_file, data_sobel_x_image, p_rows, p_cols);
    outputImg(vertical_sobel_file, data_sobel_y_image, p_rows, p_cols);

    hipFree(d_rgb_image);
    hipFree(d_grey_image);
    hipFree(d_sobel_x);
    hipFree(d_sobel_y);

    clock_t end = clock();
    double elapsed_secs = double(end - begin);
    fflush(stdout);
    // printf("Time used: %ls",elapsed_secs);
    // printf("Error %d: %s.\n", id, errors[id]);
    // std::cout << "Error " << id << ": " << errors[id] << "." << std::endl;
    std::cout <<"Time used: " << elapsed_secs << std::endl;
	return 0;
}
